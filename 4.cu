
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <ctime>
using namespace std;

__global__ void func(unsigned long long int *a)
{ 

}

int main(void){
    int device_count; 
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&device_count);

    double seconds, t1, t2;
  
    unsigned long long int N = 131072000;
    unsigned long long int *dev_a, *hst_a;
    unsigned long long int size = sizeof( unsigned long long int );

    double sizeMByte = (double)(N * size)/pow(1024,3);
    cout << "block size =  " << (N * sizeof(hst_a))/pow(1024,2) << " MB\n" << endl;
    cout << device_count << " CUDA device(s) found \n" << endl;

    for (int device = 0; device < device_count; device++) {
        hipGetDeviceProperties(&deviceProp, device);
        cout << "GPU " << device << " " << deviceProp.name << endl << endl;

        ///************ Copying Host -> Device *****************
        t1 = clock();
        cout << "Copying Host -> Device " << endl;  
        hipHostMalloc((void**)&hst_a, N * size, hipHostMallocDefault);   
        hipMalloc((void**)&dev_a, N * size); 

        hipMemcpy(dev_a, hst_a, N * size, hipMemcpyHostToDevice);
        func<<< 1, 1 >>>(dev_a);

        hipDeviceSynchronize(); 
        t2 = clock();
        seconds = (double)(t2-t1)/CLOCKS_PER_SEC + 0.00000000001;
        printf("Average bandwidth: %.6f GB/s \n\n", (double)sizeMByte/seconds);
        hipFree( dev_a ); hipFree( hst_a );

        ///************ Copying Device -> Host *****************
        t1 = clock();
        cout << "Copying Device -> Host " << endl;    
        hipHostMalloc((void**)&hst_a, N * size, hipHostMallocDefault);   
        hipMalloc((void**)&dev_a, N * size); 

        hipMemcpy(hst_a, dev_a, N * size, hipMemcpyDeviceToHost);
        func<<< 1, 1 >>>(dev_a);

        hipDeviceSynchronize(); 
        t2 = clock();
        seconds = (double)(t2-t1)/CLOCKS_PER_SEC + 0.00000000001;
        printf("Average bandwidth: %.6f GB/s \n\n", (double)sizeMByte/seconds);
        hipFree( dev_a ); hipFree( hst_a );


        ///************ Copying Host -> Host *****************
        t1 = clock();
        cout << "Copying Host -> Host " << endl;      
        hipHostMalloc((void**)&hst_a, N * size, hipHostMallocDefault);  
        hipHostMalloc((void**)&dev_a, N * size, hipHostMallocDefault);  

        hipMemcpy(hst_a, dev_a, N * size, hipMemcpyHostToHost);
        func<<< 1, 1 >>>(dev_a);

        hipDeviceSynchronize(); 
        t2 = clock();
        seconds = (double)(t2-t1)/CLOCKS_PER_SEC + 0.00000000001;
        printf("Average bandwidth: %.6f GB/s \n\n", (double)sizeMByte/seconds);
        hipFree( dev_a ); hipFree( hst_a );
        

        ///************ Copying Device -> Device *****************  
        t1 = clock();
        cout << "Copying Device -> Device " << endl;  
        hipMalloc((void**)&dev_a, N * size);
        hipMalloc((void**)&hst_a, N * size);        

        hipMemcpy(hst_a, dev_a, N * size, hipMemcpyDeviceToDevice);
        func<<< 1, 1 >>>(dev_a);
        func<<< 1, 1 >>>(hst_a);

        hipDeviceSynchronize(); 
        t2 = clock();
        seconds = (double)(t2-t1)/CLOCKS_PER_SEC + 0.00000000001;
        printf("Average bandwidth: %.6f GB/s \n\n", (double)sizeMByte/seconds);
        hipFree( dev_a ); hipFree( hst_a );

        

        // PAGELOCKED
        ///************ Copying Host -> Device *****************

        t1 = clock();
        cout << "Copying Host -> Device (usnig pagelocked)" << endl;  
        hst_a = (unsigned long long int*)malloc(N * size);   
        hipMalloc((void**)&dev_a, N * size); 

        hipMemcpy(dev_a, hst_a, N * size, hipMemcpyHostToDevice);
        func<<< 1, 1 >>>(dev_a);

        hipDeviceSynchronize(); 
        t2 = clock();
        seconds = (double)(t2-t1)/CLOCKS_PER_SEC + 0.00000000001;
        printf("Average bandwidth: %.6f GB/s \n\n", (double)sizeMByte/seconds);
        hipFree( dev_a ); free( hst_a );



        ///************ Copying Device -> Host *****************
        t1 = clock();
        cout << "Copying Device -> Host (usnig pagelocked)" << endl; 
        hst_a = (unsigned long long int*)malloc(N * size); 
        hipMalloc((void**)&dev_a, N * size); 

        hipMemcpy(hst_a, dev_a, N * size, hipMemcpyDeviceToHost);
        func<<< 1, 1 >>>(dev_a);

        hipDeviceSynchronize(); 
        t2 = clock();
        seconds = (double)(t2-t1)/CLOCKS_PER_SEC + 0.00000000001;
        printf("Average bandwidth: %.6f GB/s \n\n", (double)sizeMByte/seconds);
        hipFree( dev_a ); free( hst_a );
        
    }
    return 0;
}