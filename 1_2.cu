
#include <hip/hip_runtime.h>
#include<iostream> 
using namespace std;
int main()
{
	int device_count;
	hipGetDeviceCount(&device_count);
	hipDeviceProp_t dp;
	cout << "CUDA device count: " << device_count << "\n";
	for (int i = 0; i < device_count; i++)
	{
		hipGetDeviceProperties(&dp, i);
		cout << i << ": " << dp.name << " with CUDA compute compatibility " << dp.major << "." << dp.minor << "\n";
	}
	return 0;
}
