#include "hip/hip_runtime.h"
//�������� ���� �������� �� GPU 
#include <iostream> 
using namespace std;
#define N 10 //������������ ��� ���������� ��������� � ��������, � ��� ���������� ����� ��� GPU
__global__ void add(int *a, int *b, int *c)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}
int main(void)
{
	int a[N], b[N], c[N];
	int *dev_a,  *dev_b,  *dev_c;
	int numThreadsPerBlock = 10;
	int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

	//�������� ������ �� GPU ��� ������� a,b,c//hipMalloc �������� ������ �� ���������� ������, �.�. �� ����������.
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	//��������� ������� 'a' � 'b' �� CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
		b[i] = i + 1;
	}
	//�������� ������� 'a' � 'b' �� GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	//��������� ���� �� N ������
	add << <numBlocks, numThreadsPerBlock >> > (dev_a, dev_b, dev_c);
	// �������� ������ 'c' � GPU �� CPU
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	//������� ���������
	for (int i = 0; i < N; i++){
		cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	//����������� ������, ���������� �� GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
