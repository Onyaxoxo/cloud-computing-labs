#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/random.h>
using namespace std;

#define N 10000
__device__ int Max_Of(int a, int b, int c) {
	int max;
	int mas[3] = { a,b,c };
	max = mas[0];
	for (int i = 0; i < 3; i++) {
		if (max < mas[i]) {
			max = mas[i]; // ����� �������������
		}
	}
	return max;
}
__global__ void add(int *a, int *b, int *c, int *d) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	thrust::default_random_engine rand(123 * tid);
	thrust::uniform_int_distribution<int> dist(0, 9);
	a[tid] = dist(rand);
	b[tid] = dist(rand);
	c[tid] = dist(rand);

	if (tid < N) {
		d[tid] = Max_Of(a[tid], b[tid], c[tid]);
	}
}

int main(void) {
	int a[N], b[N], c[N], d[N];
	int *dev_a,  *dev_b,  *dev_c,  *dev_d;
	int numThreadsPerBlock = 1023; // ���� (1023 ��������
	int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock; // �����

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	hipMalloc((void**)&dev_d, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice); // �������� ������ � ��� �� ���
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);

	add << <numBlocks, numThreadsPerBlock >> > (dev_a, dev_b, dev_c, dev_d); // ���� �����

	hipMemcpy(d, dev_d, N * sizeof(int), hipMemcpyDeviceToHost); // �������� � ��� �� ���
	hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 5; i++) {
		cout << "a|" << a[i] << "		" << "b|" << b[i] << "		" << "c|" << c[i] << "		" << "d|" << d[i] << endl;
	}

	cout << " " << endl;

	for (int i = N - 5; i < N; i++) {
		cout << "a|" << a[i] << "		" << "b|" << b[i] << "		" << "c|" << c[i] << "		" << "d|" << d[i] << endl;
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
