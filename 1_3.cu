#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using namespace std;

int main(void)
{
    hipDeviceProp_t dp;
    int dev_count;

    hipGetDeviceCount(&dev_count);

    cout << dev_count << " CUDA devices" << "\n";

    //int i = 0;
    for (int i = 0; i < dev_count; ++i)
    {
        hipGetDeviceProperties(&dp, i);

        cout << i << "\n";
        cout << dp.name << "\n";
        cout << "Clock frequency " << dp.clockRate << "KHz" << "\n";

        cout << "Global Memory " << dp.totalGlobalMem << "b" << "\n";
        cout << "Global Constant " << dp.totalConstMem << "b" << "\n";
        cout << "Maximum pitch " << dp.memPitch << "b" << "\n";

        cout << "Number of multiprocessors " << dp.multiProcessorCount << "\n";
        cout << "Shared memory available per block " << dp.sharedMemPerBlock << "\n";
        cout << "Registers available per block  " << dp.regsPerBlock << "\n";
        cout << "Warp size in threads " << dp.warpSize << "\n";
        cout << "Maximum number of threads per block " << dp.maxThreadsPerBlock << "\n";
        cout << "Maximum size of each dimension of a block  " << dp.maxThreadsDim[0] << " " << dp.maxThreadsDim[1] << " " << dp.maxThreadsDim[2] << "\n";
        cout << "Maximum size of each dimension of a grid " << dp.maxGridSize[0] << " " << dp.maxGridSize[1] << " " << dp.maxGridSize[2] << endl;

    }
    return 0;
}
