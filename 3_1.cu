#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

//������ ��� ������ ������
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		cout << hipGetErrorString(err) << " in file '" << file << "' at line " << line << endl;
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define N 10
__global__ void add(int* a, int* b, int* c)
{
	//���������� ������, ����������� �� ����� �������
	int tid = blockIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}
int main(void) {
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;
	//�������� ������ �� GPU ��� ������� a,b,c
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));
	//��������� ������� 'a' � 'b' �� CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	//�������� ������� 'a' � 'b' �� GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	//��������� ���� �� N ������
	add << <N, 1 >> > (dev_a, dev_b, dev_c);
	//�������� ������ 'c' � GPU �� CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
	//������� ���������
	for (int i = 0; i < N; i++)
	{
		cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	//����������� ������, ���������� �� GPU
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));
	return 0;
}