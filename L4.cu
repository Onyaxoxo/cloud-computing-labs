#include <iostream>
#include <ctime>
#include <stdio.h>

using namespace std;

#define N 1000000000

int main(void) {
	int Host1[N], Host2[N];
	for (int i = 0; i < N; i++) {
		Host1[i] = i * i;
	}
	int* Device1, * Device2;
	hipMalloc((void**)&Device1, N * sizeof(int));
	hipMalloc((void**)&Device2, N * sizeof(int));

	std::time_t time = std::time(NULL);
	hipMemcpy(Host2, Host1, N * sizeof(int), hipMemcpyHostToHost);	   // HostToHost
	hipDeviceSynchronize();
	std::time_t time_HtH = std::time(NULL) - time;

	time = std::time(NULL);
	hipMemcpy(Device1, Host1, N * sizeof(int), hipMemcpyHostToDevice);  // HostToDevice 
	hipDeviceSynchronize();
	std::time_t time_HtD1 = std::time(NULL) - time;

	hipHostMalloc((void**)&Host1, N * sizeof(int));	                       // HostToDevice pagelocking
	time = std::time(NULL);
	hipMemcpy(Device1, Host1, N * sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	std::time_t time_HtD2 = std::time(NULL) - time;

	time = std::time(NULL);
	hipMemcpy(Host2, Device2, N * sizeof(int), hipMemcpyDeviceToHost); // DeviceToHost 
	hipDeviceSynchronize();
	std::time_t time_DtH1 = std::time(NULL) - time;

	hipHostMalloc((void**)&Host2, N * sizeof(int));	                       // DeviceToHost pagelocking
	time = std::time(NULL);
	hipMemcpy(Host2, Device2, N * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::time_t time_DtH2 = std::time(NULL) - time;

	time = std::time(NULL);
	hipMemcpy(Device2, Device1, N * sizeof(int), hipMemcpyDeviceToDevice);           // DeviceToDevice
	hipDeviceSynchronize();
	std::time_t time_DtD = std::time(NULL) - time;

	cout << "Host1 - " << (N * sizeof(int) / (1024 ^ 3)) << " GB" << endl;

	printf("Bandwidth:\n");
	printf("HostToHost: %d\n", time_HtH);
	printf("HostToDevice normal: %d\n", time_HtD1);
	printf("HostToDevice pagelocking: %d\n", time_HtD2);
	printf("DeviceToHost normal: %d\n", time_DtH1);
	printf("DeviceToHost pagelocking: %d\n", time_DtH2);
	printf("DeviceToDevice: %d\n", time_DtD);

	hipFree(Device1);
	hipFree(Device2);
	return 0;
}
