#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/random.h>
using namespace std;
//������ ��� ������ ������
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		cout << hipGetErrorString(err) << " in file '" << file << "' at line " << line << endl;
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
// ����� ������������� �������� ��������� ��������.
#define N 10000
__device__ int max_of(int a, int b, int c) {
	int max;
	int mas[3] = { a,b,c };
	max = mas[0];
	for (int i = 0; i < 3; i++) {
		if (max < mas[i]) max = mas[i];
	}
	return max;
}
__global__ void add(int* a, int* b, int* c, int* d) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	thrust::default_random_engine rand(123 * tid);
	thrust::uniform_int_distribution<int> dist(0, 9);

	a[tid] = dist(rand);
	b[tid] = dist(rand);
	c[tid] = dist(rand);

	if (tid < N) {
		d[tid] = max_of(a[tid], b[tid], c[tid]);
	}
}

int main(void) {
	int a[N], b[N], c[N], d[N];
	int* dev_a, * dev_b, * dev_c, * dev_d;
	int numThreadsPerBlock = 1023;
	int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_d, N * sizeof(int)));

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

	add << <numBlocks, numThreadsPerBlock >> > (dev_a, dev_b, dev_c, dev_d);

	HANDLE_ERROR(hipMemcpy(d, dev_d, N * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(b, dev_b, N * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for(int i = 0; i < 5; i++) {
		cout << "a[" << a[i] << "		" << "b["<< b[i] << "		" << "c[" << c[i] << "		" << "d[" << d[i] << endl;
	}

	cout << " " << endl;

	for(int i = N - 5; i < N; i++) {
		cout << "a[" << a[i] << "		" << "b[" << b[i] << "		" << "c[" << c[i] << "		" << "d[" << d[i] << endl;
	}

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));

	return 0;
}