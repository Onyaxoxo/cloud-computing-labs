#include "hip/hip_runtime.h"
//�������� ���� �������� �� GPU 
#include <iostream> 
using namespace std;
#define N 10 //������������ ��� ���������� ��������� � ��������, � ��� ���������� ����� ��� GPU
__global__ void add(int* a, int* b, int* c)
{

	int tid = blockIdx.x; //���������� ������, ����������� �� ����� ������� �����

	//threadIdx - ���������� ���� � ����� ����� (threadIdx.x, threadIdx.y, threadIdx.z), �������� �� ������
	//��������� 1023
	//blockIdx - ���������� ����� ����� � ����� (blockIdx.x, blockIdx.y, blockIdx.z), �������� �� ������
	// ��������� 65535 �� ������ �� ���������
	//blockDim - ������� ����� ����� (blockDim.x, blockDim.y, blockDim.z)
	//gridDim - ������� ����� ������ ����� (gridDim.x, gridDim.y, gridDim.z)

	if (tid < N)
		c[tid] = a[tid] + b[tid];
}
int main(void)
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;
	//�������� ������ �� GPU ��� ������� a,b,c//hipMalloc �������� ������ �� ���������� ������, �.�. �� ����������.
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	//��������� ������� 'a' � 'b' �� CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
		b[i] = i + 1;
	}
	//�������� ������� 'a' � 'b' �� GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	//��������� ���� �� N ������
	add << <N, 1 >> > (dev_a, dev_b, dev_c);
	// �������� ������ 'c' � GPU �� CPU
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	//������� ���������
	for (int i = 0; i < N; i++)
	{
		cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	//����������� ������, ���������� �� GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
